#include "hip/hip_runtime.h"
/*
 * simulate.c
 *
 * Implement your (parallel) simulation here!
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>

#include "timer.h"
#include "file.h"
#include "simulate.h"


using namespace std;

#define MAX_BLOCK_SIZE 512

/* Copied from vector-add.cu */
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

 /* This we run on a GPU */
__global__ void calculate_next(double *dev_old, double *dev_cur,
        double *dev_new, int i_max, int timestep) {

    /* Define timesteps for */
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    unsigned int t_id = threadIdx.x;
    
    if (i < i_max) {
     __shared__ double current[MAX_BLOCK_SIZE];
     
     current[t_id] = dev_cur[i];
     __syncthreads();
     
     /* The formula from Assign 2_1 */
     if (t_id == 0){
      dev_new[i] = 2 * current[t_id] - dev_old[i] + 0.2 *
      (dev_cur[i - 1] - (2 * current[t_id] - current[t_id + 1]));
     }else if (t_id == blockDim.x - 1) {
      dev_new[i] = 2 * current[t_id] - dev_old[i] + 0.2 *
      (current[t_id - 1] - (2 * current[t_id] - dev_cur[i + 1]));
     }else {
      dev_new[i] = 2 * current[t_id] - dev_old[i] + 0.2 *
      (current[t_id - 1] - (2 * current[t_id] - current[t_id + 1]));
     }
    }
}

/* SIMULATOR FROM ASSIGNMENT 2_1 FRAMEWORK */
double *simulate(const int i_max, const int t_max, const int num_threads,
        double *old_array, double *current_array, double *next_array){
   double *dev_old, *dev_cur, *dev_new;

    /* Allocate data (as shown in vector-add.cu, ln 20-25) */
    checkCudaCall(hipMalloc(&dev_old, i_max * sizeof(double)));
    checkCudaCall(hipMalloc(&dev_cur, i_max * sizeof(double)));
    checkCudaCall(hipMalloc(&dev_new, i_max * sizeof(double)));

    /* Start CUDA events (vector-add.cu, ln 60-63) */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* Copy Vector Data (vector-add.cu, ln 65) */
    checkCudaCall(hipMemcpy(dev_old, old_array, i_max * sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(dev_cur, current_array, i_max * sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(dev_new, next_array, i_max * sizeof(double), hipMemcpyHostToDevice));

    /* Start Timer (vector-add.cu) */
    hipEventRecord(start, 0);
    
    /* Repeat for Number of Threads */
    for (int t = 1; t < t_max; t++) {
        /* See function definition above,
         * since we did this in a loop in the previous assignments
         */
        calculate_next<<<ceil((double)i_max/num_threads), num_threads>>>(
                dev_old, dev_cur, dev_new, i_max - 1, t);
        
        /* Do the Switcharoo */
        double *temp = dev_old;
        dev_old = dev_cur;
        dev_cur = dev_new;
        dev_new = temp;
    }

   /* Stop Timer (vector-add.cu) */
   hipEventRecord(stop, 0);
    
   /* Check for Errors (vector-add.cu) */
    checkCudaCall(hipGetLastError());

   /* Copy back results (vector-add.cu) */
   checkCudaCall(hipMemcpy(current_array, dev_cur, i_max * sizeof(double), hipMemcpyDeviceToHost));
    
   /* Free (vector-add.cu) */
   checkCudaCall(hipFree(dev_old));
   checkCudaCall(hipFree(dev_cur));
   checkCudaCall(hipFree(dev_new));

   /* You should return a pointer to the array with the final results. */
   float elapsedTime;
   hipEventElapsedTime(&elapsedTime, start, stop);

   printf("time spent in kernel: %f miliseconds\n", elapsedTime);
   return current_array;
}

typedef double (*func_t)(double x);

/* Copied from Assignment 2_1 (with minor modifications) */
double gauss(double x)
{
    return exp((-1 * x * x) / 2);
}

/* Copied from Assignment 2_1 (with minor modifications) */
void fill(double *array, int offset, int range, double sample_start,
        double sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}

/* Copied from Assignment 2_1 (with minor modifications) */
int main(int argc, char *argv[])
{
    double *old, *current, *next, *ret;
    int t_max, i_max, num_threads;
    timer vectorAddTimer("vector add timer");


    /* Parse commandline args: i_max t_max num_threads */
    if (argc < 4) {
        printf("Usage: %s i_max t_max num_threads [initial_data]\n", argv[0]);
        printf(" - i_max: number of discrete amplitude points, should be >2\n");
        printf(" - t_max: number of discrete timesteps, should be >=1\n");
        printf(" - num_threads: number of threads to use for simulation, "
                "should be >=1\n");
        printf(" - initial_data: select what data should be used for the first "
                "two generation.\n");
        printf("   Available options are:\n");
        printf("    * sin: one period of the sinus function at the start.\n");
        printf("    * sinfull: entire data is filled with the sinus.\n");
        printf("    * gauss: a single gauss-function at the start.\n");
        printf("    * file <2 filenames>: allows you to specify a file with on "
                "each line a float for both generations.\n");

        return EXIT_FAILURE;
    }

    i_max = atoi(argv[1]);
    t_max = atoi(argv[2]);
    num_threads = atoi(argv[3]);

    if (i_max < 3) {
        printf("argument error: i_max should be >2.\n");
        return EXIT_FAILURE;
    }
    if (t_max < 1) {
        printf("argument error: t_max should be >=1.\n");
        return EXIT_FAILURE;
    }
    if (num_threads < 1) {
        printf("argument error: num_threads should be >=1.\n");
        return EXIT_FAILURE;
    }

    /* Allocate and initialize buffers. */
    old = (double *) malloc(i_max * sizeof(double));
    current = (double *) malloc(i_max * sizeof(double));
    next = (double *) malloc(i_max * sizeof(double));

    if (old == NULL || current == NULL || next == NULL) {
        fprintf(stderr, "Could not allocate enough memory, aborting.\n");
        return EXIT_FAILURE;
    }

    memset(old, 0, i_max * sizeof(double));
    memset(current, 0, i_max * sizeof(double));
    memset(next, 0, i_max * sizeof(double));

    /* How should we will our first two generations? */
    if (argc > 4) {
        if (strcmp(argv[4], "sin") == 0) {
            fill(old, 1, i_max/4, 0, 2*3.14, sin);
            fill(current, 2, i_max/4, 0, 2*3.14, sin);
        } else if (strcmp(argv[4], "sinfull") == 0) {
            fill(old, 1, i_max-2, 0, 10*3.14, sin);
            fill(current, 2, i_max-3, 0, 10*3.14, sin);
        } else if (strcmp(argv[4], "gauss") == 0) {
            fill(old, 1, i_max/4, -3, 3, gauss);
            fill(current, 2, i_max/4, -3, 3, gauss);
        } else if (strcmp(argv[4], "file") == 0) {
            if (argc < 7) {
                printf("No files specified!\n");
                return EXIT_FAILURE;
            }
            file_read_double_array(argv[5], old, i_max);
            file_read_double_array(argv[6], current, i_max);
        } else {
            printf("Unknown initial mode: %s.\n", argv[4]);
            return EXIT_FAILURE;
        }
    } else {
        /* Default to sinus. */
        fill(old, 1, i_max/4, 0, 2*3.14, sin);
        fill(current, 2, i_max/4, 0, 2*3.14, sin);
    }

    /* Minor Modifications */
    vectorAddTimer.start();

    ret = simulate(i_max, t_max, num_threads, old, current, next);
    
    vectorAddTimer.stop();

    cout << vectorAddTimer;

    file_write_double_array("result.txt", ret, i_max);
    /* End Minor Modifications */
    
    free(old);
    free(current);
    free(next);

    return EXIT_SUCCESS;
}
