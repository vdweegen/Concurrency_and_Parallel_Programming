#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "timer.h"
#include <iostream>

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

__global__ void vectorReduceKernel(int n, float* deviceA, float* deviceResult){
   unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
   
    __shared__ double sf[1024];
    sf[threadIdx.x] = deviceA[index];
    __syncthreads();
    
    // do comparison, copy biggest to result array
    if ((index % 2 == 0) && index < n) {
        if (sf[index] > sf[index+1]){
            deviceResult[index/2] = sf[index];
        }else{
            deviceResult[index/2] = sf[index+1];    
        }
    }
}


void vectorAddCuda(int n, float* a, float* result) {
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        cout << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));

    // execute kernel
    hipEventRecord(start, 0);
    vectorReduceKernel<<<n/threadBlockSize, threadBlockSize>>>(n, deviceA, deviceResult);
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceResult));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;
}


int main(int argc, char* argv[]) {
    int n = 1024;
    timer vectorAddTimer("vector add timer");
    float* a = new float[n];
    float* result = new float[n];

    // initialize the vectors.
    for(int i=0; i<n; i++) {
        a[i] = (float)rand()/(float)RAND_MAX;
    }

    vectorAddTimer.start();
    vectorAddCuda(n, a, result);
    vectorAddTimer.stop();

    cout << vectorAddTimer;

    // verify the resuls
    for(int i=0; i<n; i++) {
        if(result[i] != 2*i) {
            cout << "error in results! Element " << i << " is " << result[i] << ", but should be " << (2*i) << endl;
            exit(1);
        }
    }
    cout << "results OK!" << endl;
            
    delete[] a;
    delete[] result;
    
    return 0;
}
